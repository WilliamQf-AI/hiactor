#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

//Kernel
__global__ void d_vec_add(int *d_a, int *d_b, int *d_c,int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        d_c[i] = d_a[i] + d_b[i];
}

extern "C"
void* d_alloc_space(unsigned num_bytes) {
    void *ret;
    hipMalloc(&ret, num_bytes);
    return ret;
}

extern "C"
void d_free_space(void *ptr) {
    hipFree(ptr);
}

extern "C" 
void h_vec_add(stream_handle strm_hdl, int *a, int *b, int *c, int *d_a, int *d_b, int *d_c, unsigned n) {
    hipMemcpyAsync(d_a, a, sizeof(int) * n, hipMemcpyHostToDevice, static_cast<hipStream_t>(strm_hdl));
    hipMemcpyAsync(d_b, b, sizeof(int) * n, hipMemcpyHostToDevice, static_cast<hipStream_t>(strm_hdl));

    dim3 DimGrid(n / BX + 1, 1, 1);
    dim3 DimBlock(BX, 1, 1);
    d_vec_add<<<DimGrid, DimBlock, 0, static_cast<hipStream_t>(strm_hdl)>>>(d_a, d_b, d_c, n);

    hipMemcpyAsync(c, d_c, sizeof(int) * n, hipMemcpyDeviceToHost, static_cast<hipStream_t>(strm_hdl));
}
