/** Copyright 2021 Alibaba Group Holding Limited. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "device.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C"
void cuda_set_device(uint32_t dev_id){
    hipSetDevice(dev_id);
}

extern "C"
uint32_t cuda_get_device(){
    int32_t dev_id;
    hipGetDevice(&dev_id);
    return dev_id;
}

extern "C"
uint32_t cuda_get_device_count(){
    int32_t dev_cnt = 0;
    hipGetDeviceCount(&dev_cnt);
    return dev_cnt;
}

extern "C"
cuda_stream_handle create_cuda_stream() {
    hipStream_t retval;
    hipStreamCreate(&retval);
    return static_cast<cuda_stream_handle>(retval);
}

extern "C"
cuda_event_handle create_cuda_event() {
    hipEvent_t stop;
    hipEventCreate(&stop);
    return static_cast<cuda_event_handle>(stop);
}

extern "C" 
void cuda_stream_event_record(cuda_stream_handle strm_hdl, cuda_event_handle eve_hdl){
    hipEventRecord(static_cast<hipEvent_t>(eve_hdl), static_cast<hipStream_t>(strm_hdl));
}

extern "C" 
bool cuda_check(cuda_event_handle eve_hdl){
    return !(hipEventQuery(static_cast<hipEvent_t>(eve_hdl)) == hipErrorNotReady);
}
